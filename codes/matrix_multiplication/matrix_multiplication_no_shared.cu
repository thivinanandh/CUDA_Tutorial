// Author : Thivin Anandh
// Purpose : Baselevel CUDA kernel matrix multiplication code without shared memory

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


__global__ void matrix_multiplication(double *A, double* B, double* C, int N)
{
    // Get the the thread parameters
    int row = threadIdx.x + blockIdx.x * blockDim.x;

    // for each row perform the computations
    if (row < N)
    {
        // loop over j
        for (int j = 0 ; j < N ; j++)
        {
            C[row*N + j] = 0.0;
            for (int k = 0; k < N ; k++)
            {
                C[row*N + j] += A[row*N + k] * B[N*k + j];
            }
        }
    }
}

void matrix_multiplication_host(double *A, double* B, double* C, int N)
{
    // loop over i
    for (int i = 0 ; i < N ; i++)
    {
        // loop over j
        for (int j = 0 ; j < N ; j++)
        {
            C[i*N + j] = 0.0;
            for (int k = 0; k < N ; k++)
            {
                C[i*N + j] += A[i*N + k] * B[k*N + j];
            }
        }
    }
}



// main function
int main(int argc, char** argv)
{
    // Allocate double array for storing matrix
    double *host_A, *host_B, *host_C;
    double *device_A, *device_B, *device_C;

    // Matrix size
    int N = 1000;

    //Allocate memory on host
    host_A = (double*)malloc(N*N*sizeof(double));
    host_B = (double*)malloc(N*N*sizeof(double));
    host_C = (double*)malloc(N*N*sizeof(double));

    // Initialize matrix
    for(int i=0; i<N; i++)
    {
        for(int j=0; j<N; j++)
        {
            host_A[i*N+j] = i-j + 3;
            host_B[i*N+j] = i+j - 2;
        }
    }

    // Allocate memory on device
    hipMalloc(&device_A, N*N*sizeof(double));
    hipMalloc(&device_B, N*N*sizeof(double));
    hipMalloc(&device_C, N*N*sizeof(double));

    // Copy data from host to device
    hipMemcpy(device_A, host_A, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_B, host_B, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_C, host_C, N*N*sizeof(double), hipMemcpyHostToDevice);

    // lets setup the grid and block size
    int num_threads = 256;
    int thread_block_size = ceil(N*N/num_threads);

    // Start the timer
    hipEvent_t start, stop;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch kernel
    matrix_multiplication<<<thread_block_size, num_threads>>>(device_A, device_B, device_C, N);


    // Stop the timer
    hipEventRecord(stop);

    // Synchronize the events
    hipEventSynchronize(stop);

    // Calculate the time
    float milliseconds_gpu = 0;
    hipEventElapsedTime(&milliseconds_gpu, start, stop);

    printf("Time taken for matrix multiplication is %f ms\n", milliseconds_gpu);

    // wait for the kernel to finish
    hipDeviceSynchronize();

    // transfer the array to host
    hipMemcpy(host_C, device_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

    // create a new array for validation of answr
    double* C_check = (double*)malloc(N*N*sizeof(double));

    // time the host code
    hipEventRecord(start);

    // callhost matrix multiplication routines
    matrix_multiplication_host(host_A, host_B, host_C, N);

    // Stop the timer
    hipEventRecord(stop);

    // Synchronize the events
    hipEventSynchronize(stop);

    // Calculate the time
    float milliseconds_host = 0;
    hipEventElapsedTime(&milliseconds_host, start, stop);

    printf("Time taken for matrix multiplication on host is %f ms\n", milliseconds_host);

    // print the speedup
    printf("Speedup is %f\n", milliseconds_host/milliseconds_gpu);


    // check the results
    for(int i=0; i<N; i++)
    {
        for(int j=0; j<N; j++)
        {
            if (host_C[i*N+j] != host_C[i*N+j])
            {
                printf("Error in the results\n");
                return 1;
            }
        }
    }

    return 0;
}